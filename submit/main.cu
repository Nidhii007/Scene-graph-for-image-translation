#include "hip/hip_runtime.h"
	/*
		CS 6023 Assignment 3. 
		Do not make any changes to the boiler plate code or the other files in the folder.
		Use hipFree to deallocate any memory not in usage.
		Optimize as much as possible.
	*/

	#include "SceneNode.h"
	#include <queue>
	#include "Renderer.h"
	#include <stdio.h>
	#include <string.h>
	#include <hip/hip_runtime.h>
	#include <chrono>

	__global__
	void scenemake2(int offset, int xdir, int ydir, int amount, int *dGlobalCoordinatesX, int *dGlobalCoordinatesY, int *dCsr, int *dOffset, int numOfChild){
		int id=blockIdx.x*blockDim.x+threadIdx.x;
		if(id<numOfChild){
			id+=offset;
			atomicAdd(&dGlobalCoordinatesX[dCsr[id]],xdir*amount);
			atomicAdd(&dGlobalCoordinatesY[dCsr[id]],ydir*amount);
			int numOfThreads=dOffset[dCsr[id]+1]-dOffset[dCsr[id]];
			int numberOfBlocks = (numOfThreads+blockDim.x-1)/blockDim.x;
			scenemake2<<<numberOfBlocks, numOfThreads>>>(dOffset[dCsr[id]], xdir, ydir, amount,dGlobalCoordinatesX, dGlobalCoordinatesY, dCsr, dOffset,numOfThreads);
		}
	}
	__global__ void scenemake(int meshNum , int xdir, int ydir, int amount,int* dGlobalCoordinatesX,int* dGlobalCoordinatesY,int* dCsr,int* dOffset, int numOfChild){
		int numOfThreads=dOffset[meshNum+1]-dOffset[meshNum];
		int numberOfBlocks=(numOfThreads+blockDim.x-1)/blockDim.x;
		if(ydir==0)	atomicAdd(&dGlobalCoordinatesX[meshNum],xdir*amount);
		else 	atomicAdd(&dGlobalCoordinatesY[meshNum],ydir*amount);
		scenemake2<<<numberOfBlocks,numOfThreads>>>(dOffset[meshNum],xdir,ydir,amount,dGlobalCoordinatesX,dGlobalCoordinatesY,dCsr,dOffset,numOfThreads);    
	}
	__global__
	void renderx(int *mesh_i,int dGlobalX_i, int dGlobalY_i, int dOpacity_i, int *dframeOp,int frameSizeX, int frameSizeY, int *dFinalPng){
		int mesh_ind=blockIdx.x*blockDim.x+threadIdx.x;
		int inFrame_x=dGlobalX_i+blockIdx.x;
		int inFrame_y=dGlobalY_i+threadIdx.x;
		int inFrame_ind=inFrame_x*frameSizeY+inFrame_y;
		if(inFrame_x>=0 && inFrame_y>=0 && inFrame_y<frameSizeY && inFrame_x<frameSizeX && dframeOp[inFrame_ind]<dOpacity_i){
			dFinalPng[inFrame_ind]=mesh_i[mesh_ind];
			dframeOp[inFrame_ind]=dOpacity_i;
		}
	}


	void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
		/* Function for parsing input file*/

		FILE *inputFile = NULL;
		// Read the file for input. 
		if ((inputFile = fopen (fileName, "r")) == NULL) {
			printf ("Failed at opening the file %s\n", fileName) ;
			return ;
		}

		// Input the header information.
		int numMeshes ;
		fscanf (inputFile, "%d", &numMeshes) ;
		fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;
		

		// Input all meshes and store them inside a vector.
		int meshX, meshY ;
		int globalPositionX, globalPositionY; // top left corner of the matrix.
		int opacity ;
		int* currMesh ;
		for (int i=0; i<numMeshes; i++) {
			fscanf (inputFile, "%d %d", &meshX, &meshY) ;
			fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
			fscanf (inputFile, "%d", &opacity) ;
			currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
			for (int j=0; j<meshX; j++) {
				for (int k=0; k<meshY; k++) {
					fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
				}
			}
			//Create a Scene out of the mesh.
			SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ; 
			scenes.push_back (scene) ;
		}

		// Input all relations and store them in edges.
		int relations;
		fscanf (inputFile, "%d", &relations) ;
		int u, v ; 
		for (int i=0; i<relations; i++) {
			fscanf (inputFile, "%d %d", &u, &v) ;
			edges.push_back ({u,v}) ;
		}

		// Input all translations.
		int numTranslations ;
		fscanf (inputFile, "%d", &numTranslations) ;
		std::vector<int> command (3, 0) ;
		for (int i=0; i<numTranslations; i++) {
			fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
			translations.push_back (command) ;
		}
	}


	void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
		/* Function for writing the final png into a file.*/
		FILE *outputFile = NULL; 
		if ((outputFile = fopen (outputFileName, "w")) == NULL) {
			printf ("Failed while opening output file\n") ;
		}
		
		for (int i=0; i<frameSizeX; i++) {
			for (int j=0; j<frameSizeY; j++) {
				fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
			}
			fprintf (outputFile, "\n") ;
		}
	}


	int main (int argc, char **argv) {
		
		// Read the scenes into memory from File.
		const char *inputFileName = argv[1] ;
		int* hFinalPng ; 

		int frameSizeX, frameSizeY ;
		std::vector<SceneNode*> scenes ;
		std::vector<std::vector<int> > edges ;
		std::vector<std::vector<int> > translations ;
		readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
		hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;
		
		// Make the scene graph from the matrices.
		Renderer* scene = new Renderer(scenes, edges) ;

		// Basic information.
		int V = scenes.size () ;
		int E = edges.size () ;
		int numTranslations = translations.size () ;

		// Convert the scene graph into a csr.
		scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
		int *hOffset = scene->get_h_offset () ;  
		int *hCsr = scene->get_h_csr ();
		int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
		int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
		int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
		int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
		int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
		int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

		auto start = std::chrono::high_resolution_clock::now () ;
		
		// Code begins here.
		// Do not change anything above this comment.
		
		memset(hFinalPng, 0, sizeof (int) * frameSizeX * frameSizeY);

		int *dOffset;
		int *dCsr;
		// int *dOpacity;
		// int **dMesh;
		int *dGlobalCoordinatesX;
		int *dGlobalCoordinatesY;
		// int *dFrameSizeX;
		// int *dFrameSizeY;
		int *mesh_i;
		int *dFinalPng;

		hipMalloc(&dOffset, sizeof (int) * (V+1)) ;
		hipMalloc(&dGlobalCoordinatesX, sizeof (int) * V) ;
		hipMalloc(&dGlobalCoordinatesY, sizeof (int) * V) ;
		hipMalloc(&dCsr, sizeof (int) * E) ;
		hipMalloc(&dFinalPng, sizeof (int) * frameSizeX * frameSizeY);
		
		hipMemcpy(dOffset, hOffset, sizeof (int) * (V+1), hipMemcpyHostToDevice);
		hipMemcpy(dGlobalCoordinatesX, hGlobalCoordinatesX, sizeof (int) * V, hipMemcpyHostToDevice);
		hipMemcpy(dGlobalCoordinatesY, hGlobalCoordinatesY, sizeof (int) * V, hipMemcpyHostToDevice);
		hipMemcpy(dCsr, hCsr, sizeof (int) * E, hipMemcpyHostToDevice);
		
		int xdir, ydir;
		for(int i=0;i<numTranslations;i++){
			int meshNum=translations[i][0];
			int command=translations[i][1];
			int amount=translations[i][2];
			if(command==0){//up
				xdir=-1;
				ydir=0;
			}
			else if(command==1){//down
				xdir=1;
				ydir=0;
			}
			else if(command==2){//left
				xdir=0;
				ydir=-1;
			}
			else{//right
				xdir=0;
				ydir=1;
			}
			scenemake<<<1,1>>>(meshNum, xdir, ydir, amount,dGlobalCoordinatesX, dGlobalCoordinatesY, dCsr, dOffset, 1);
		}	
		hipDeviceSynchronize();5
		hipFree(dOffset);
		hipFree(dCsr);

		hipMemcpy(hGlobalCoordinatesX, dGlobalCoordinatesX, sizeof (int) * V, hipMemcpyDeviceToHost);
		hipMemcpy(hGlobalCoordinatesY, dGlobalCoordinatesY, sizeof (int) * V, hipMemcpyDeviceToHost);
		
		hipFree(dGlobalCoordinatesX);
		hipFree(dGlobalCoordinatesY);
		
		int *hframeOp = (int*) malloc (sizeof(int) * frameSizeX * frameSizeY) ;
		memset(hframeOp, INT_MIN, sizeof(int) * frameSizeX * frameSizeY);
		
		int *dframeOp;
		hipMalloc(&dframeOp , sizeof(int) * frameSizeX * frameSizeY) ;
		hipMemcpy(dframeOp,hframeOp,sizeof(int) * frameSizeX * frameSizeY,hipMemcpyHostToDevice);
		
		hipMemcpy(dFinalPng, hFinalPng, sizeof (int) * frameSizeX * frameSizeY,hipMemcpyHostToDevice);
		
		hipMalloc(&mesh_i,10000*sizeof(int));
		for(int i=0;i<V;i++){
			hipMemcpy(mesh_i, hMesh[i], sizeof (int) * hFrameSizeX[i]*hFrameSizeY[i], hipMemcpyHostToDevice);
			renderx<<<hFrameSizeX[i],hFrameSizeY[i]>>>(mesh_i, hGlobalCoordinatesX[i], hGlobalCoordinatesY[i], hOpacity[i], dframeOp, frameSizeX, frameSizeY, dFinalPng);
		}
		hipDeviceSynchronize();

		hipMemcpy(hFinalPng, dFinalPng, sizeof (int) * frameSizeX * frameSizeY, hipMemcpyDeviceToHost);

		// for(int i=0;i<frameSizeX;i++){
		// 	for(int j=0;j<frameSizeY;j++){
		// 		printf("%d\t",hFinalPng[i*frameSizeY+j]);
		// 	}
		// 	printf("\n");
		// }
		

		// Do not change anything below this comment.
		// Code ends here.

		auto end  = std::chrono::high_resolution_clock::now () ;

		std::chrono::duration<double, std::micro> timeTaken = end-start;

		printf ("execution time : %f\n", timeTaken) ;
		// Write output matrix to file.
		const char *outputFileName = argv[2] ;
		writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;	

	}
